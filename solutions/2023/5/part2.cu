// Part 2 but CUDA and its a complete brute force search
// this took ~800ms... this is insane


#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>

using namespace std;

__global__ void findSeedLocKernel(int64_t seedStart, int64_t range, int64_t **maps, int *maplengths, int64_t *outArr)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int64_t minLocFound = INT64_MAX;
  for (int64_t i = seedStart + index; i < range + seedStart; i += stride)
  {
    int64_t loc = i;
    for (int j = 0; j < 7; j++)
    {
      for (int k = 0; k < maplengths[j]; k++)
      {
        int64_t to = maps[j][k * 3];
        int64_t from = maps[j][k * 3 + 1];
        int64_t range = maps[j][k * 3 + 2];

        if (loc >= from && loc <= from + range - 1)
        {
          loc = to + (loc - from);
          break;
        }
      }
    }

    if (loc < minLocFound)
    {
      minLocFound = loc;
    }
  }

  if (minLocFound < INT64_MAX)
    outArr[index] = minLocFound;
}

int main()
{
  ifstream infile("inputeasy.txt");

  int numSeedPairs;
  infile >> numSeedPairs;

  vector<pair<int64_t, int64_t>> seedPairs;
  for (int i = 0; i < numSeedPairs; i++)
  {
    int64_t x, y;
    infile >> x >> y;
    seedPairs.push_back(make_pair(x, y));
  }

  int numMaps = 7;
  int *mapLengths;
  hipMallocManaged(&mapLengths, numMaps * sizeof(int));

  // create an array of arrays in cuda memory
  int64_t **maps;
  hipMallocManaged(&maps, numMaps * sizeof(int64_t *));

  infile >> numMaps;

  for (int i = 0; i < numMaps; i++)
  {
    int mapLength;
    infile >> mapLength;
    mapLengths[i] = mapLength;

    int64_t *map;
    hipMallocManaged(&map, mapLength * sizeof(int64_t) * 3);
    for (int j = 0; j < mapLength; j++)
    {
      // maps come in a triplet

      infile >> map[j * 3];
      infile >> map[j * 3 + 1];
      infile >> map[j * 3 + 2];
    }
    maps[i] = map;
  }

  int64_t totMin = INT64_MAX;
  for (auto seedPair : seedPairs)
  {
    int64_t seedStart = seedPair.first;
    int64_t seedEnd = seedPair.second;

    int64_t range = seedEnd - seedStart + 1;

    int blockSize = 512;
    int numBlocks = 200;

    int64_t *outArr;
    // each thread will write to a different index in the outArr
    hipMallocManaged(&outArr, blockSize * numBlocks * sizeof(int64_t));

    findSeedLocKernel<<<numBlocks, blockSize>>>(seedStart, range, maps, mapLengths, outArr);
    hipDeviceSynchronize();

    // find the min of the outArr
    int64_t minLocFound = INT64_MAX;
    for (int i = 0; i < blockSize * numBlocks; i++)
    {
      if (outArr[i] < minLocFound && outArr[i] != 0)
      {
        minLocFound = outArr[i];
      }
    }

    if (minLocFound < totMin)
    {
      totMin = minLocFound;
    }

    // free the memory
    hipFree(outArr);
  }

  // free the memory
  for (int i = 0; i < numMaps; i++)
  {
    hipFree(maps[i]);
  }

  hipFree(maps);
  hipFree(mapLengths);

  cout << totMin << endl;
}